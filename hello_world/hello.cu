#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>


__global__ void cuda_hello(){
  printf("Hello world from GPU\n");
}

int main(int argc, char** argv){
  cuda_hello<<<1,1>>>();
  hipDeviceSynchronize();
  return 0;
}
